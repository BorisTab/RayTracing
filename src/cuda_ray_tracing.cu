#include "hip/hip_runtime.h"
#include <cuda_ray_tracing.h>

__global__ void Cuda::Cuda_canvas_intersect(uchar3* d_canvas, Scene& scene, Sphere* spheres, size_t spheres_size, double double_max, Light* lights, size_t lights_size) {
    size_t y = blockDim.y * blockIdx.y + threadIdx.y;
    size_t x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x >= scene._canvas._width || y >= scene._canvas._height) {
        return;
    }

    printf("%lud %lu\n", x, y);

    size_t pixel_num = y * scene._canvas._width + x;

    auto ray_to_pixel = SceneF::Ray_to_pixel_from_camera(scene, x, y);
    double min_dist = double_max;
    size_t min_dist_sphere_num = 0;
    Vector3<double> normal;
    Material sphere_material;

    SphereF::Scene_intersect(spheres, spheres_size, scene._camera_pos, ray_to_pixel, min_dist, min_dist_sphere_num, normal, sphere_material);
    min_dist = double_max;

    printf("%lu", pixel_num);
//    copy_simple_color(SphereF::Run_ray(spheres[min_dist_sphere_num], scene._camera_pos, Vec3::normalize(ray_to_pixel), spheres, spheres_size, scene, lights, lights_size), d_canvas[pixel_num]);
//    d_canvas[pixel_num] = SphereF::Run_ray(spheres[min_dist_sphere_num], scene._camera_pos, Vec3::normalize(ray_to_pixel), spheres, spheres_size, scene, lights, lights_size);
}