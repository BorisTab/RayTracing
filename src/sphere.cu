#include "hip/hip_runtime.h"
#include <sphere.h>
#include <cuda_error_checker.h>
#include <cuda_ray_tracing.h>

#include <iostream>
#include <limits>
#include <future>

#define DIST_MAX 1000000000000000000000000000.

__device__ uchar3 SphereF::Run_ray(Sphere& sphere, const Vector3<double> &origin, const Vector3<double> &ray, Sphere* spheres, size_t spheres_size, const Scene& scene, const Light* lights, size_t lights_size, size_t depth) {
    Vector3<double> normal;
    double diffuse_light_intensity = 0;
    double specular_light_intensity = 0;
    size_t min_dist_sphere_num = 0;
    double dist = DIST_MAX;
    Material intersect_material;

    if (depth <= 4 && Scene_intersect(spheres, spheres_size, origin, ray, dist, min_dist_sphere_num, normal, intersect_material)) {
        auto dir_from_camera_to_point = Vec3::normalize(ray) * dist;
        auto dir_to_point = origin + dir_from_camera_to_point;

        auto reflected_ray_dir = Vec3::reflect(Vec3::normalize(ray), Vec3::normalize(normal));
        auto refracted_ray_dir = Vec3::refract(Vec3::normalize(ray), Vec3::normalize(normal), intersect_material.refract_index);

        auto reflected_ray_origin = reflected_ray_dir * normal < 0 ?  dir_to_point - normal * 1e-3 : dir_to_point + normal * 1e-3;
        auto refracted_ray_origin = refracted_ray_dir * normal < 0 ?  dir_to_point - normal * 1e-3 : dir_to_point + normal * 1e-3;

        uchar3 reflected_color = Run_ray(sphere, reflected_ray_origin, reflected_ray_dir, spheres, spheres_size, scene, lights, lights_size, depth + 1);
        uchar3 refracted_color = Run_ray(sphere, refracted_ray_origin, refracted_ray_dir, spheres, spheres_size, scene, lights, lights_size, depth + 1);

        for (int light_num = 0; light_num < lights_size; ++light_num) {
            auto light_dir_from_point_to_light = (lights[light_num].position - dir_to_point);
            double light_dist = Vec3::len(light_dir_from_point_to_light);
            Vec3::normalize_self(light_dir_from_point_to_light);

            auto shadow_origin = light_dir_from_point_to_light * normal < 0 ?  dir_to_point - normal * 1e-3 : dir_to_point + normal * 1e-3;

            double min_shadow_dist = DIST_MAX;
            Vector3<double> shadow_normal;
            Material shad_mat;

            if (Scene_intersect(spheres, spheres_size, shadow_origin, light_dir_from_point_to_light, min_shadow_dist, min_dist_sphere_num, shadow_normal, shad_mat) && min_shadow_dist < light_dist) {
                continue;
            }

            diffuse_light_intensity += lights[light_num].intensity * max(0., light_dir_from_point_to_light * Vec3::normalize(normal));
            specular_light_intensity += lights[light_num].intensity * pow(max(0., Vec3::reflect(light_dir_from_point_to_light, normal) * ray), intersect_material.specular_power);
        }

        uchar3 white_color = {255, 255, 255};
//        SimpleColor out_color = intersect_material.diffuse_color;

        return white_color;
//        return out_color * intersect_material.albedo[0] +
//               out_color * diffuse_light_intensity * intersect_material.albedo[1] +
//               white_color * specular_light_intensity * intersect_material.albedo[2] +
//               reflected_color * intersect_material.reflectivity +
//               refracted_color * intersect_material.refractivity;
    }

    uchar3 white_color = {255, 255, 255};
    return white_color;//SceneF::Background_pixel(scene, ray);
}

//void Sphere

__device__ bool SphereF::Ray_intersect(Sphere& sphere, const Vector3<double> &origin, const Vector3<double>& ray, double& dist_to_sphere, Vector3<double>& normal) {
    Vector3<double> line_to_center = sphere.center - origin;
    double distance_from_center_to_ray = Vec3::parallelogram_area(ray, line_to_center) / Vec3::len(ray);
    double min_dist = 0;

    if (distance_from_center_to_ray <= sphere.radius) {
//        if (line_to_center.cos(ray) < 0) {
//            return false;
//        }

        min_dist = Vec3::len(line_to_center) * (Vec3::normalize(line_to_center) * Vec3::normalize(ray)) -
                sqrt(sphere.radius * sphere.radius - distance_from_center_to_ray * distance_from_center_to_ray);
        if (min_dist < 0) {
            return false;
        }

        dist_to_sphere = min_dist;
        Vec3::copy_vec(Vec3::normalize(ray) * dist_to_sphere - line_to_center, normal);
    }
    else {
        distance_from_center_to_ray = DIST_MAX;
    }

    return distance_from_center_to_ray < sphere.radius;
}

//bool SphereF::Ray_intersect(Sphere& sphere, const Vector3<double> &origin, const Vector3<double>& ray, double& dist_to_sphere) {
//    Vector3<double> line_to_center = sphere.center - origin;
//
//    double distance_from_center_to_ray = ray.parallelogram_area(line_to_center) / ray.len();
//
//    if (distance_from_center_to_ray <= sphere.radius) {
//        if (line_to_center.cos(ray) < 0) {
//            return false;
//        }
//        dist_to_sphere = line_to_center.len() * line_to_center.cos(ray) -
//                         sqrt(sphere.radius * sphere.radius - distance_from_center_to_ray * distance_from_center_to_ray);
//    }
//    else {
//        distance_from_center_to_ray = std::numeric_limits<double>::max();
//    }
//
//    return distance_from_center_to_ray < sphere.radius;
//}

__device__ bool SphereF::Scene_intersect(Sphere* spheres, size_t spheres_size, const Vector3<double>& origin, const Vector3<double>& ray_to_pixel, double& min_dist, size_t& min_dist_sphere_num, Vector3<double>& normal, Material& intersect_material) {
    for (size_t sphere_num = 0; sphere_num < spheres_size; ++sphere_num) {
        double dist = 0;
        Vector3<double> cur_normal;

        if (Ray_intersect(spheres[sphere_num], origin, ray_to_pixel, dist, cur_normal) && dist < min_dist) {
            min_dist = dist;
            min_dist_sphere_num = sphere_num;

            Vec3::copy_vec(cur_normal, normal);
            intersect_material = spheres[sphere_num].material;
        }
    }

    if (fabs(ray_to_pixel.y) > 1e-3)  {
        double plane_dist = -(origin.y-4)/ray_to_pixel.y; // the checkerboard plane has equation y = -4
        Vector3<double> pt = origin + ray_to_pixel * plane_dist;
        if (plane_dist > 0 && fabs(pt.x) < 10 && pt.z > 10 && pt.z < 30 && plane_dist < min_dist) {
            min_dist = plane_dist;

            Vec3::set_val(normal, 0., -1., 0.);

            intersect_material.albedo[0] = 0.3;
            intersect_material.albedo[1] = 1;

            SimpleColor white = {255, 255, 255};
            SimpleColor yellow = {255, 179, 76};

            intersect_material.diffuse_color = (int(.5 * pt.x + 1000) + int(.5 * pt.z)) & 1 ? white : yellow;
        }
    }

    return min_dist < 1000000;
}

void SphereF::Set_spheres_on_scene(Scene &scene, std::vector<Sphere> &spheres) {
    auto start = std::clock();

    checkCudaErrors(hipSetDevice(0));

    auto* pixels = scene._canvas.pixels;
    size_t num_of_pixels = scene._canvas._height * scene._canvas._width;

    uchar3* d_canvas = nullptr;
    checkCudaErrors(hipMalloc(&d_canvas, sizeof(uchar3) * num_of_pixels));
    checkCudaErrors(hipMemcpy(d_canvas, pixels, sizeof(uchar3) * num_of_pixels, hipMemcpyHostToDevice));

    size_t threadNum = 32;
    dim3 blockSize(threadNum, 1, 1);
    dim3 gridSize(scene._canvas._width / threadNum + 1, scene._canvas._height, 1);

    Cuda::Cuda_canvas_intersect<<<gridSize, blockSize>>>(d_canvas, scene, &spheres.front(), spheres.size(), DIST_MAX, &scene._lights.front(), scene._lights.size());
    checkCudaErrors(hipGetLastError());

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

//    checkCudaErrors(hipMemcpy(pixels, d_canvas, sizeof(SimpleColor) * num_of_pixels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_canvas));

    auto time = (std::clock() - start) / (double) CLOCKS_PER_SEC;
    printf("%g\n", time);
}
