#include "hip/hip_runtime.h"
#include <scene.h>

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>
#undef STB_IMAGE_IMPLEMENTATION

__device__ Light::Light(const Vector3<double> &position, double intensity):
    position(position.x, -position.y, position.z),
    intensity(intensity) {}

void SceneF::Set_canvas(Scene& scene, size_t height, size_t width, const Color &bg_color) {
    CanvasF::Setup(scene._canvas, height, width, bg_color);
}

void SceneF::Set_camera(Scene& scene, const Vector3<double> &camera_pos, double fov) {
    scene._camera_pos = camera_pos;
    scene._fov = fov;
}

void SceneF::Set_lights(Scene& scene, const std::vector<Light> &lights) {
    scene._lights = lights;
}

void SceneF::Set_background_pic(Scene& scene, const char* filepath, int desired_channels) {
    int width = 0;
    int height = 0;
    int channels_in_file = 0;
    unsigned char* buffer = stbi_load(filepath, &width, &height, &channels_in_file, desired_channels);

    printf("channels in envmap file: %d, size: %d x %d\n", channels_in_file, width, height);

    scene._background_pic.resize(height);
    int pixel_pos_in_buffer = 0;
    for (int line_num = 0; line_num < height; ++line_num) {
        scene._background_pic[line_num].resize(width);

        for (int pixel_pos = 0; pixel_pos < width; ++pixel_pos) {
            pixel_pos_in_buffer += 3;
            scene._background_pic[line_num][pixel_pos] = Color(buffer[pixel_pos_in_buffer], buffer[pixel_pos_in_buffer + 1], buffer[pixel_pos_in_buffer + 2]);
        }
    }

    scene._background_pic_height = height;
    scene._background_pic_width = width;

    stbi_image_free(buffer);
}

//std::vector<std::vector<Color>> & Scene::Get_background_pic() {
//    return _background_pic;
//}
//
//const std::vector<Light>& Scene::Get_lights() {
//    return _lights;
//}

__device__ Vector3<double> SceneF::Ray_to_pixel_from_camera(Scene& scene, size_t x, size_t y) {
    double z_pos = scene._camera_pos.z + 1;

    double canvas_height = 2 * z_pos * std::tan(scene._fov / 2.);
    double canvas_width = canvas_height * scene._canvas._width / (scene._canvas._height * 1.);

    double x_pos = ((x + 0.5) / (scene._canvas._width  * 1.) - 0.5) * canvas_width;
    double y_pos = ((y + 0.5) / (scene._canvas._width * 1.) - 0.5) * canvas_height;

    Vector3<double> ray_to_pixel = {x_pos, y_pos, z_pos};

    return ray_to_pixel;
}

//Canvas& Scene::Get_canvas() {
//    return _canvas;
//}
//
//const Vector3<double> & Scene::Get_camera_pos() {
//    return _camera_pos;
//}

int SceneF::Save_canvas_to_png(Scene& scene, const char* filepath) {
    return CanvasF::Save_to_png(scene._canvas, filepath);
}

__device__ SimpleColor SceneF::Background_pixel(const Scene& scene, const Vector3<double> &ray_to_pixel) {
    Vector3<double> z_vector = {0, 0, 1};
    Vector3<double> y_vector = {0, 1, 0};

    Vector3<double> xz_vector = {ray_to_pixel.x, 0, ray_to_pixel.z};
    Vector3<double> yz_vector = {0, ray_to_pixel.y, ray_to_pixel.z};

    size_t bg_x = acos(Vec3::normalize(xz_vector) * z_vector) / M_PI *
            scene._background_pic_width / 2 * (ray_to_pixel.x / fabs(ray_to_pixel.x)) +
            scene._background_pic_width / 2;

    size_t bg_y = (1 - acos(Vec3::normalize(yz_vector) * y_vector) / M_PI) *
            static_cast<double>(scene._background_pic_height);

    return scene._background_pic[std::min(bg_y, scene._background_pic_height - 1)][std::min(bg_x, scene._background_pic_width - 1)];
}
